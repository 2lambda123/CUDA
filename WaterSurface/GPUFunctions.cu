#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <hip/hip_runtime.h> 

__global__ void updateWaveMapGPU2( float* dev_newWave, float* dev_oldWave,unsigned int *dev_arraySize ){
		int tid= blockIdx.x;
		while (tid < *dev_arraySize){
			unsigned int x = int(tid / 8000);
			unsigned int y = tid % 8000;
			unsigned int up = (x - 1) + (y * 8000);
			unsigned int down = (x + 1) + (y * 8000);
			unsigned int leftp = x + ((y - 1)* 8000);
			unsigned int rightp = x + ((y + 1)* 8000);


			float n = 0;
			int no=0;
			if (x-1 >= 0) {
				n += dev_oldWave[up]; 
				no++;
			}
			if (x + 1 < 8000) {
				n += dev_oldWave[down]; 
				no++;
			}
			if (y-1 >= 0) {
				n += dev_oldWave[leftp]; 
				no++;
			}
			if (y+1 < 8000) {
				no++;
				n += dev_oldWave[rightp]; 
			}
			
			n /= no;
			//n = n*2 - dev_newWave[tid];
			n = n - (n/32.0f);
			dev_newWave[tid] = n;
			tid += gridDim.x;
		}
}

void updateWaveMapGPU1(float* dev_newWave, float* dev_oldWave,unsigned int *dev_arraySize){
	updateWaveMapGPU2<<< 1 ,1 >>>(dev_newWave,dev_oldWave,dev_arraySize);
}

